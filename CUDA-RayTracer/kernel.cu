#include "hip/hip_runtime.h"
﻿#define STB_IMAGE_WRITE_IMPLEMENTATION

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#include ""

#include <stdio.h>
#include <iostream>
#include <thrust/device_vector.h>

#define CUDA_VERSION 11030;
//#define GLM_FORCE_CUDA

#include "../middleware/glm/glm/glm.hpp"
#include "../middleware/stb/stb_image_write.h"

#include "Utilities/CudaHelpers.h"
#include "Utilities/Ray.h"
#include "Utilities/ShadeRec.h"
#include "Utilities/Random.h"

#include "Cameras/ThinLensCamera.h"
#include "Cameras/PinholeCamera.h"

#include "Materials/Material.h"
#include "Materials/CookTorrence.h"

#include "Lights/Light.h"
#include "Lights/AmbientLight.h"
#include "Lights/AmbientOccluder.h"
#include "Lights/PointLight.h"
#include "Lights/AreaLight.h"

#include "Scene/Scene.h"

#include "Tracers/Whitted.h"
#include "Tracers/PathTrace.h"
#include "Tracers/BranchPathTrace.h"

#include "GeometricObjects/GeometricObj.h"
#include "GeometricObjects/Instance.h"
#include "GeometricObjects/Sphere.h"
#include "GeometricObjects/Plane.h"
#include "GeometricObjects/Rectangle.h"
#include "GeometricObjects/Triangle.h"
#include "GeometricObjects/Ellipse.h"
#include "GeometricObjects/Torus.h"

#include "Acceleration/BVHAccel.h"

#include "GeometricObjects/Compound/Model.h"

using namespace glm;

const int SCR_WIDTH = 1024;
const int SCR_HEIGHT = 1024;

//using Rectangle;

__global__ void render_ThinLensCamera(float3* fb, Scene** scene_ptr, ThinLensCamera** camera_ptr) {
    float3		pixel_color = make_float3(0,0,0);
    ViewPlane   vp((*scene_ptr)->vp);
    Ray			ray;
    float2		sp;				// sample point in [0, 1] x [0, 1]
    float2		pp;				// sample point on a pixel
    float2		dp;				// sample point on unit disk
    float2		lp;				// sample point on lens
    
    vp.s = (*camera_ptr)->get_zoom();
    
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int j = threadIdx.y + blockIdx.y * blockDim.y;
    //unsigned long long threadId = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

    if ((i >= vp.hres) || (j >= vp.vres)) return;

    int pixel_index = j * vp.hres + i;
    
    float lens_radius = (*camera_ptr)->get_lens_radius();
    Sampler* sampler_ptr = (*camera_ptr)->get_sampler();

    
    for (int n = 0; n < 32; n++) {
        sp = UniformSampleSquare();
        //sp = vp.sampler_ptr->sample_unit_square();
     
        pp.x = vp.s * (i - 0.5 * vp.hres + sp.x);
        pp.y = vp.s * (j - 0.5 * vp.vres + sp.y);
        
        //dp = sampler_ptr->sample_unit_disk();
        dp = ConcentricSampleDisk();
        lp = dp * lens_radius;
   
        ray.o = (*camera_ptr)->position + lp.x * (*camera_ptr)->right + lp.y * (*camera_ptr)->up;
        ray.d = (*camera_ptr)->ray_direction(pp, lp);

        pixel_color += (*scene_ptr)->tracer_ptr->trace_ray(ray);
    }

    pixel_color /= 32;
    pixel_color *= (*camera_ptr)->exposure_time;

    pixel_color /= (pixel_color + 1.0f); // Hard coded Reinhard tone mapping

    if (vp.gamma != 1.f)
        pixel_color = pow(pixel_color, vp.inv_gamma);
    
    //unsigned int seed = (i * 1024 + j) * 100;
    
    //fb[pixel_index] = float3(wang_hash(seed) * (1.0 / 4294967296.0));//pixel_color;
    //fb[pixel_index] = float3(sampler_ptr->sample_unit_disk(), 0.f);
    fb[pixel_index] = pixel_color;
}

__global__ void create_ThinLensCamera(ThinLensCamera** camera_ptr)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        (*camera_ptr) = new ThinLensCamera(make_float3(0, 200, 1000), make_float3(0, 70, 0));
        //(*camera_ptr) = new ThinLensCamera(float3(150, 100, 370), float3(0, 20, 0));
        //(*camera_ptr) = new ThinLensCamera(float3(-300, 200, 370), float3(0, 15, 0));
        (*camera_ptr)->exposure_time = 1.f;
        (*camera_ptr)->set_view_distance(100);
        (*camera_ptr)->set_zoom(30);
        (*camera_ptr)->set_sampler(new MultiJittered(1));
        (*camera_ptr)->set_lens_radius(15.f);
        (*camera_ptr)->set_focal_distance(900.f);
        (*camera_ptr)->update_camera_vectors();
    }
}

__global__ void setup_random_states(hiprandState_t states[]) 
{
    int id = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    hiprand_init(clock(), id, 0, &states[id]);
}

__global__ void create_scene(Scene** scene_ptr, Triangle* triangles, int num_triangles, LinearBVHNode* __restrict__ nodes, int ordered_prims[], hiprandState_t states[])
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {

        (*scene_ptr) = new Scene;

        (*scene_ptr)->states = states;

        ViewPlane* vp = new ViewPlane;
        vp->set_hres(SCR_WIDTH);
        vp->set_vres(SCR_WIDTH);
        vp->set_samples(1);
        vp->set_pixel_size(1.f);
        vp->set_gamma(1.f);
        vp->set_max_depth(1);
        (*scene_ptr)->vp = (*vp);

        (*scene_ptr)->set_tracer(new PathTrace((*scene_ptr)));

        MultiJittered* sampler_ptr = new MultiJittered(1);
        sampler_ptr->generate_samples();
        sampler_ptr->map_to_hemisphere(1);
        
        AmbientLight* ambient_ptr = new AmbientLight();
        //ambient_ptr->set_sampler(sampler_ptr);
        (*scene_ptr)->ambient_ptr = ambient_ptr;
        
        int number_lights = 1;
        (*scene_ptr)->lights = CudaList<Light*>(number_lights);

        //PointLight* light_ptr = new PointLight();
        //light_ptr->set_position(100, 500, 500);
        //light_ptr->scale_radiance(10);
        //(*scene_ptr)->add_light(light_ptr);

        //int number_objects = 10;
        int number_objects = num_triangles;
        (*scene_ptr)->objects = CudaList<GeometricObj*>(number_objects);

        #pragma region Cornell Box

        // BOTTOM //
        
        CookTorrence* material_ptr = new CookTorrence();
        material_ptr->set_cd(make_float3(1,1,1));
        material_ptr->set_ka(0.05f);
        material_ptr->set_kd(0.f);
        material_ptr->set_ks(1.f);
        material_ptr->set_roughness(0.1);
        material_ptr->set_diffuse_sampler(sampler_ptr);

        Plane* plane_ptr = new Plane(make_float3(0, -20, 0), make_float3(0.f, 1.f, 0.f));
        plane_ptr->set_material(material_ptr);
        //(*scene_ptr)->add_obj(plane_ptr);

        // RIGHT //

        material_ptr = new CookTorrence();
        material_ptr->set_cd(make_float3(1.f, 0.2f, 0.2f));
        material_ptr->set_ka(0.05f);
        material_ptr->set_kd(1.f);
        material_ptr->set_ks(0.f);
        material_ptr->set_roughness(100.f);
        material_ptr->set_diffuse_sampler(sampler_ptr);

        Rectangle* rect_ptr = new Rectangle(make_float3(-125, -22, 125), make_float3(0, 0, -250), make_float3(0, 250, 0), make_float3(1, 0, 0));
        //rect_ptr->set_sampler(sampler_ptr);
        //rect_ptr->enable_shadows(false);
        rect_ptr->set_material(material_ptr);
        //(*scene_ptr)->add_obj(rect_ptr);
        /*
        Emissive* emissive_ptr = new Emissive;
        emissive_ptr->scale_radiance(20);
        emissive_ptr->set_ce(1, 0, 0);

        AreaLight* area_light_ptr = new AreaLight;
        area_light_ptr->set_object(rect_ptr);
        area_light_ptr->set_material(emissive_ptr);
        area_light_ptr->enable_shadows(true);

        (*scene_ptr)->add_light(area_light_ptr);
        */
        // LEFT //

        material_ptr = new CookTorrence();
        material_ptr->set_cd(make_float3(0.2f, 0.2f, 1.f));
        material_ptr->set_ka(0.05f);
        material_ptr->set_kd(1.f);
        material_ptr->set_ks(0.f);
        material_ptr->set_roughness(100.f);
        material_ptr->set_diffuse_sampler(sampler_ptr);

        rect_ptr = new Rectangle(make_float3(125, -22, 125), make_float3(0, 0, -250), make_float3(0, 250, 0), make_float3(-1, 0, 0));
        //rect_ptr->set_sampler(sampler_ptr);
        //rect_ptr->enable_shadows(false);
        rect_ptr->set_material(material_ptr);
        //(*scene_ptr)->add_obj(rect_ptr);
        /*
        emissive_ptr = new Emissive;
        emissive_ptr->scale_radiance(20);
        emissive_ptr->set_ce(0,0,1);

        area_light_ptr = new AreaLight;
        area_light_ptr->set_object(rect_ptr);
        area_light_ptr->set_material(emissive_ptr);
        area_light_ptr->enable_shadows(true);

        (*scene_ptr)->add_light(area_light_ptr);
        */
        // TOP //

        material_ptr = new CookTorrence();
        material_ptr->set_cd(make_float3(1,1,1));
        material_ptr->set_ka(0.05f);
        material_ptr->set_kd(1.f);
        material_ptr->set_ks(0.f);
        material_ptr->set_roughness(100.f);
        material_ptr->set_diffuse_sampler(sampler_ptr);

        rect_ptr = new Rectangle(make_float3(-125, 228, -125), make_float3(250, 0, 0), make_float3(0, 0, 250), make_float3(0, -1, 0));
        rect_ptr->set_material(material_ptr);
        //(*scene_ptr)->add_obj(rect_ptr);

        // BACK //
        
        material_ptr = new CookTorrence();
        material_ptr->set_cd(make_float3(1,1,1));
        material_ptr->set_ka(0.05f);
        material_ptr->set_kd(1.f);
        material_ptr->set_ks(0.f);
        material_ptr->set_roughness(100.f);
        material_ptr->set_diffuse_sampler(sampler_ptr);

        rect_ptr = new Rectangle(make_float3(-125, -22, -125), make_float3(250, 0, 0), make_float3(0, 250, 0), make_float3(0, 0, 1));
        rect_ptr->set_material(material_ptr);
        //(*scene_ptr)->add_obj(rect_ptr);
        
        // Light //

        Emissive* emissive_ptr = new Emissive;
        emissive_ptr->scale_radiance(250);
        emissive_ptr->set_ce(1,1,1);
        //emissive_ptr->set_ce(0.96470, 0.80392, 0.54509);

        Sphere* blah = new Sphere(make_float3(0, 225, 0), 25);
        blah->set_material(emissive_ptr);
        blah->set_sampler(new MultiJittered(1));
        blah->enable_shadows(false);

        rect_ptr = new Rectangle(make_float3(-37.5, 386.74999, 0), make_float3(75, 0, 0), make_float3(0, 0, 75), make_float3(0, -1, 0));
        //rect_ptr = new Rectangle(dfloat3(-125, 227.9999, -125), dfloat3(250, 0, 0), dfloat3(0, 0, 250), dfloat3(0, -1, 0));
        rect_ptr->set_material(emissive_ptr);
        rect_ptr->set_sampler(new MultiJittered(1));
        rect_ptr->enable_shadows(false);

        AreaLight* area_light_ptr = new AreaLight;
        area_light_ptr->set_object(rect_ptr);
        area_light_ptr->set_material(emissive_ptr);
        area_light_ptr->enable_shadows(true);

        //(*scene_ptr)->add_obj(rect_ptr);
        (*scene_ptr)->add_light(area_light_ptr);

        material_ptr = new CookTorrence();
        material_ptr->set_cd(make_float3(0,0,0));
        material_ptr->set_ka(0.05f);
        material_ptr->set_kd(1.f);
        material_ptr->set_ks(1.f);
        material_ptr->set_roughness(0.1);
        material_ptr->set_diffuse_sampler(sampler_ptr);

        Sphere* sphere_ptr = new Sphere(make_float3(0, 30, 0), 50.f);
        sphere_ptr->set_material(material_ptr);
        //(*scene_ptr)->add_obj(sphere_ptr);

        material_ptr = new CookTorrence();
        material_ptr->set_cd(make_float3(1,0.2,0.2));
        material_ptr->set_ka(0.05f);
        material_ptr->set_kd(1.f);
        material_ptr->set_ks(1.f);
        material_ptr->set_roughness(100.f);
        material_ptr->set_diffuse_sampler(sampler_ptr);

        sphere_ptr = new Sphere(make_float3(50, 0, 100), 20.f);
        sphere_ptr->set_material(material_ptr);
        //(*scene_ptr)->add_obj(sphere_ptr);

        material_ptr = new CookTorrence();
        material_ptr->set_cd(make_float3(0.2, 1, 0.2));
        material_ptr->set_ka(0.05f);
        material_ptr->set_kd(1.f);
        material_ptr->set_ks(1.f);
        material_ptr->set_roughness(100.f);
        material_ptr->set_diffuse_sampler(sampler_ptr);

        sphere_ptr = new Sphere(make_float3(0, 0, 100), 20.f);
        sphere_ptr->set_material(material_ptr);
        //(*scene_ptr)->add_obj(sphere_ptr);

        material_ptr = new CookTorrence();
        material_ptr->set_cd(make_float3(0.2,0.2,1));
        material_ptr->set_ka(0.05f);
        material_ptr->set_kd(1.f);
        material_ptr->set_ks(1.f);
        material_ptr->set_roughness(100.f);
        material_ptr->set_diffuse_sampler(sampler_ptr);

        sphere_ptr = new Sphere(make_float3(-50, 0, 100), 20.f);
        sphere_ptr->set_material(material_ptr);
        //(*scene_ptr)->add_obj(sphere_ptr);

        #pragma endregion

        for (int i = 0; i < num_triangles; i++) {
            int index = ordered_prims[i];
            (*scene_ptr)->objects[i] = &triangles[index];
        }

        (*scene_ptr)->bvh = nodes;
    }
}

/*__global__ void foobar()
{    
    hiprandState_t state;
    hiprand_init(clock(), 0, 0, &state);

    for (int i = 0; i < 32; i++) {
        randStates[i] = hiprand_uniform(&state) * UINT32_MAX;
    }
    __syncthreads();
}*/

void save_image(const char* filename, const void* data) {
    int stride = 3 * SCR_WIDTH;
    stbi_write_png(filename, SCR_WIDTH, SCR_HEIGHT, 3, data, stride);
}

int main()
{

    int nx = SCR_WIDTH;
    int ny = SCR_HEIGHT;
    int tx = 8;
    int ty = 8;

    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);

    //size_t x = 0;
    //checkCudaErrors(cudaThreadGetLimit(&x, hipLimitStackSize));
    checkCudaErrors(cudaThreadSetLimit(hipLimitStackSize, 20000));
    checkCudaErrors(hipDeviceSetLimit(hipLimitMallocHeapSize, 5000 * 100000 * sizeof(Triangle*)));

    // tuple< h_triangles, d_triangles, list_size >//
    //std::vector<tuple<GeometricObj**, GeometricObj**, int>> primitive_merge_list;

    //unsigned int h_randStates[32];
    //for (int i = 0; i < 32; i++) {
    //    h_randStates[i] = rand();
    //}

    //checkCudaErrors(hipMalloc((void**)&h_randStates, sizeof(unsigned int) * 32));
    //checkCudaErrors(hipMalloc((void**)&randStates, sizeof(unsigned int) * 32));

    //foobar <<< 1, 1, sizeof(unsigned int) * 32 >>> ();
    //checkCudaErrors(hipGetLastError());
    //checkCudaErrors(hipDeviceSynchronize());

    std::vector<Model*> models;

    //models.push_back(new Model("E:/repos/CUDA-RayTracer/models/plane.obj"));
    //models.push_back(new Model("E:/repos/CUDA-RayTracer/models/ico-sphere.obj"));
    models.push_back(new Model("E:/repos/CUDA-RayTracer/models/dragon_test.obj"));
    //models.push_back(new Model("E:/repos/CUDA-RayTracer/models/Nefertiti.obj"));

    int nmb_triangles;
    std::vector<BVHPrimitiveInfo> triangle_info;
    Triangle* d_triangles = loadModels(models, triangle_info, nmb_triangles);

    std::cerr << "Building BVH with " << nmb_triangles << " primitives. ";
    clock_t start, stop;
    start = clock();

    BVHAccel* bvh = new BVHAccel(triangle_info, SplitMethod::SAH, 8);

    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n\n";
    
    ThinLensCamera** tlcam_ptr;
    checkCudaErrors(hipMalloc((void**)&tlcam_ptr, sizeof(ThinLensCamera*)));

    create_ThinLensCamera <<< 1, 1 >>> (tlcam_ptr);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    hiprandState_t* states;
    checkCudaErrors(hipMalloc((void**)&states, sizeof(hiprandState_t)));

    setup_random_states <<< blocks, threads >>> (states);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    
    Scene** scene;
    checkCudaErrors(hipMalloc((void**)&scene, sizeof(Scene*)));
 
    create_scene <<< 1, 1 >>> (scene, d_triangles, nmb_triangles, bvh->d_nodes, bvh->d_orderedPrims, states);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks. ";

    int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(float3);

    // allocate FB
    float3* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    // Render our buffer

    start = clock();

    render_ThinLensCamera <<<blocks, threads>>> (fb, scene, tlcam_ptr);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    stop = clock();
    timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    uint8_t* data = new uint8_t[nx * ny * 3];

    // Output FB as Image
    int index = 0;

    // Output FB as Image
    for (int j = ny - 1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j * nx + i;
            int ir = int(255.99 * fb[pixel_index].x);
            int ig = int(255.99 * fb[pixel_index].y);
            int ib = int(255.99 * fb[pixel_index].z);

            data[index++] = ir;
            data[index++] = ig;
            data[index++] = ib;
        }
    }
    checkCudaErrors(hipFree(fb));

    save_image("E:/repos/CUDA-RayTracer/images/out.png", data);
    
    return 0;
}
